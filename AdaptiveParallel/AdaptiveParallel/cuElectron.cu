#include "hip/hip_runtime.h"
#include "cuElectron.h"
#include "Electron.h"

#include "hip/hip_runtime.h"
#include ""

#include "Constants.h"

void /*cuElectron::*/cuMalloc(void **cuElectrons, int memSize)
{
    hipMalloc(cuElectrons, memSize);
}

extern "C" __global__ void pole_cuda(TPL_Point *cuElectrons, int b1, int e1, float QK, float MK)
{

	int i = blockIdx.x*blockDim.x+threadIdx.x;

	if (i<e1-b1)
	{
		i+=b1;

		float v;
		 cuElectrons[i].ax=-1*QK*cuElectrons[i].vy*B_const/MK;
		 //Y
		 if ( 
             std::abs(cuElectrons[i].z-
                                        Z1) 
             <= 
                    std::abs(cuElectrons[i].z-
                                                Z2)
             )
		 {
			 v=cos(1.57*std::abs(cuElectrons[i].z-Z1)/(Z2-((Z1+Z2)/2))); 
		 }
		 else
		 {
			 v=cos(1.57*std::abs(cuElectrons[i].z-Z2)/(Z2-((Z1+Z2)/2)));
		 }
		 cuElectrons[i].ay=(QK*cuElectrons[i].vx*B_const/MK)-(QK*E_const/MK)+(QK*E_const/MK)*std::abs(v*v*v*v*v*v*v*v*v*v*v*v*v*v*v);
		 //Z
		 if(std::abs(cuElectrons[i].z-Z1)<=std::abs(cuElectrons[i].z-Z2))
		 {
			 v=cos(1.57*std::abs(cuElectrons[i].z-Z1)/(Z2-((Z1+Z2)/2)));
			 cuElectrons[i].az=std::abs((QK*E_const/MK)*v*v*v*v*v*v*v*v*v*v*v*v*v*v*v);
		 }
		 else
		 {
			 v=cos(1.57*std::abs(cuElectrons[i].z-Z2)/(Z2-((Z1+Z2)/2)));
			 cuElectrons[i].az=(-1)*std::abs((QK*E_const/MK)*v*v*v*v*v*v*v*v*v*v*v*v*v*v*v);
		 }
	}
}


extern "C" __global__ void raschet(int *cuElectrons_number, TPL_Point *cuElectrons, int b1,int e1, int b2,int e2, int Count, float MK, float QK, int final)
{
	int i = blockIdx.x*blockDim.x+threadIdx.x;

	i*=32;

	if (i<e1-b1)
	{
		i+=b1;

		float dx,dy,dz,powe;

		if (i<final)
		for(int j=b2;j<e2;++j)
		if (j<Count)
		{
			for (int t=0;t<32;t++)
			  if(((cuElectrons_number[i+t]-Kx)<=cuElectrons_number[j])&&((cuElectrons_number[i+t]+Kx)>=cuElectrons_number[j]))
			    if((i+t!=j)&&(i+t<final))
			{
			 dx=cuElectrons[i+t].x-cuElectrons[j].x;
			 dy=cuElectrons[i+t].y-cuElectrons[j].y;
			 dz=cuElectrons[i+t].z-cuElectrons[j].z;
			 powe=(float)pow((float)(dx*dx+dy*dy+dz*dz),(float)0.5)*num_k/MK*QK*QK/(dx*dx+dy*dy+dz*dz);
			 cuElectrons[i+t].ax+=dx/powe;
			 cuElectrons[i+t].ay+=dy/powe;
			 cuElectrons[i+t].az+=dz/powe;
			 }


		 /*if(((cuElectrons_number[i]-Kx)<=cuElectrons_number[j])&&((cuElectrons_number[i]+Kx)>=cuElectrons_number[j]))
			 if(i!=j)
			 {
			 dx=cuElectrons[i].x-cuElectrons[j].x;
			 dy=cuElectrons[i].y-cuElectrons[j].y;
			 dz=cuElectrons[i].z-cuElectrons[j].z;
			 powe=(float)pow((float)(dx*dx+dy*dy+dz*dz),(float)0.5)*k/MK*QK*QK/(dx*dx+dy*dy+dz*dz);
			 cuElectrons[i].ax+=dx/powe;
			 cuElectrons[i].ay+=dy/powe;
			 cuElectrons[i].az+=dz/powe;
			 }*/



		}
	}
}

extern "C" __global__ void pereschet(int *cuElectrons_number, TPL_Point *cuElectrons, int b1, int e1, float step)
{
	int i = blockIdx.x*blockDim.x+threadIdx.x;

	if (i<e1-b1)
	{
		 i+=b1;

		 cuElectrons[i].x=cuElectrons[i].x+cuElectrons[i].vx*step+(cuElectrons[i].ax*step*step/2);
		 cuElectrons[i].vx+=cuElectrons[i].ax*step;
		 cuElectrons[i].y=cuElectrons[i].y+cuElectrons[i].vy*step+(cuElectrons[i].ay*step*step/2);
		 cuElectrons[i].vy+=cuElectrons[i].ay*step;
		 cuElectrons[i].z=cuElectrons[i].z+cuElectrons[i].vz*step+(cuElectrons[i].az*step*step/2);
		 cuElectrons[i].vz+=cuElectrons[i].az*step;
		 //������� ����� ���������
	}
}

